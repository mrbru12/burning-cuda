#include <hip/hip_runtime.h>

#include <stdio.h>

int main(int argc, char *argv[]) {
	int device_count;
	hipGetDeviceCount(&device_count);

	for (int device = 0; device < device_count; device++) {
		hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device);

		printf("pageableMemoryAccess: %d\n", prop.pageableMemoryAccess);
		printf("concurrentManagedAccess: %d\n", prop.concurrentManagedAccess);
		printf("managedMemory: %d\n", prop.managedMemory);
	}

	return 0;
}
