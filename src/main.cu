#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <SDL2/SDL.h>

#include <stdio.h>
#include <stdint.h>

#define THREADS_PER_BLOCK 256

typedef uint32_t pixel_t;

__managed__ pixel_t *pixels;

__device__ pixel_t pixel_from_rgb(int r, int g, int b) {
	pixel_t pixel;
	pixel  = b << 0;
	pixel |= g << 8;
	pixel |= r << 16;
	// pixel |= a << 24;

	return pixel;
}

__global__ void fill_pixels(pixel_t *pixels) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	pixel_t color;
	if ((threadIdx.x + threadIdx.y) % 8 == 0) {
		color = pixel_from_rgb(255, 0, 0);
	} else {
		color = pixel_from_rgb(0, 0, 0);
	}
	
	int width = gridDim.x * blockDim.x;
	pixels[y * width + x] = color;
}

// TODO: Maybe use grid_size insdead of grid_width and grid_height
void draw_grid(SDL_Surface *surface, int grid_width, int grid_height, int width, int height) {
	SDL_Rect grid_cell;
	grid_cell.w = width / grid_width;
	grid_cell.h = height / grid_height;

	for (int y = 0; y < grid_height; y++) {
		grid_cell.y = y * grid_cell.h;

		for (int x = 0; x < grid_width; x++) {
			grid_cell.x = x * grid_cell.w;

			// SDL_SetRenderDrawColor(renderer, 255, 0, 0, 255);
			// SDL_RenderDrawRect(renderer, &grid_cell);
		}
	}
}

// Create squared dims
void calculate_dims(dim3 *block_dim, dim3 *thread_dim, int width, int height) {
	int block_count = (width * height) / THREADS_PER_BLOCK;

	/*
	if (block_count % 2 != 0) {
		printf("WARN: Cannot perfectly fit the grid on the screen!\n");
	}
	*/

	int block_row = sqrt(block_count);
	int thread_row = sqrt(THREADS_PER_BLOCK);

	*block_dim = dim3(block_row, block_row);
	*thread_dim = dim3(thread_row, thread_row);
}

int main(int argc, char *argv[]) {
	if (SDL_Init(SDL_INIT_VIDEO) < 0) {
		printf("ERROR: Failed to initialize SDL!\n");
	}

	int width = 400, height = 400;
	// int grid_width = 4, grid_height = 4;

	hipMallocManaged(&pixels, width * height * sizeof(pixel_t));

	dim3 block_dim, thread_dim;
	calculate_dims(&block_dim, &thread_dim, width, height);

	fill_pixels<<<block_dim, thread_dim>>>(pixels);

	hipDeviceSynchronize();

	SDL_Surface *surface = SDL_CreateRGBSurfaceFrom(
		pixels, width, height, 32, width * sizeof(pixel_t),
		0x00FF0000, 0x0000FF00, 0x000000FF, 0
	);

	SDL_SaveBMP(surface, "/mnt/c/Users/Bruno/Desktop/out.bmp");

	SDL_FreeSurface(surface);

	/*
	int grid_width = 4, grid_height = 4;
	SDL_Texture* grid_texture = SDL_CreateTexture(
		renderer, SDL_PIXELFORMAT_RGBA8888, SDL_TEXTUREACCESS_TARGET, width, height
	);
	*/

	// SDL_DestroyTexture(grid_texture);
	// SDL_DestroyRenderer(renderer);

	hipFree(pixels);

	return 0;
}

