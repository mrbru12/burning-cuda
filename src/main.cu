#include <hip/hip_runtime.h>

#include <stdio.h>

int main(int argc, char *argv[]) {
	int device_count;
	hipGetDeviceCount(&device_count);

	for (int i = 0; i < device_count; i++) {
		hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device);

		printf("pageableMemoryAccess: %d\n", prop.pageableMemoryAccess);
		printf("concurrentManagedAccess: %d\n", prop.concurrentManagedAccess);
		printf("managedMemory: %d\n", prop.managedMemory);
	}

	return 0;
}
