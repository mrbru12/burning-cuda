#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <raylib.h>
#include <stdio.h>
#include <stdint.h>
#include <math.h>

#define THREADS_PER_BLOCK 256

__managed__ Color *pixels;

__global__ void burning_ship_kernel(Color *pixels, int width, int height, double centerX, double centerY, double scale, int max_iterations) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < width && y < height) {
        double zx = 0.0, zy = 0.0;

        double cx = centerX + ((double)x - (double)width / 2.0) * scale;
        double cy = centerY + ((double)y - (double)height / 2.0) * scale;

        int iteration = 0;
        while (zx * zx + zy * zy < 4.0 && iteration < max_iterations) {
            double temp = zx * zx - zy * zy + cx;
            zy = fabs(2.0 * zx * zy) + cy;
            zx = temp;
            iteration++;
        }

        Color color;
        if (iteration == max_iterations) {
            color = (Color){ 0, 0, 0, 255 }; // Preta para pontos fora do fractal
        } else {
            // Gradiente de cores suave com função de seno
            int r = (int)(128.0 + 127.0 * sin(0.16 * iteration + 4));
            int g = (int)(128.0 + 127.0 * sin(0.16 * iteration + 2));
            int b = (int)(128.0 + 127.0 * sin(0.16 * iteration));
            color = (Color){ r, g, b, 255 };
        }

        pixels[y * width + x] = color;
    }
}

void calculate_dims(dim3 *block_dim, dim3 *thread_dim, int width, int height) {
    int thread_row = (int)sqrt(THREADS_PER_BLOCK);
    *block_dim = dim3((width + thread_row - 1) / thread_row, (height + thread_row - 1) / thread_row);
    *thread_dim = dim3(thread_row, thread_row);
}

int main(int argc, char *argv[]) {
    const int width = 800;
    const int height = 600;

    // Inicializa a Raylib
    InitWindow(width, height, "Burning Ship Fractal - Controle Interativo");
    SetTargetFPS(60);

	SetTraceLogLevel(LOG_NONE);

    hipMallocManaged(&pixels, width * height * sizeof(Color));

    dim3 block_dim, thread_dim;
    calculate_dims(&block_dim, &thread_dim, width, height);

    // Defina as coordenadas e escala para o fractal
    double centerX = -1.761485;
    double centerY = -0.000040;
    double scale = 0.00003;
    int max_iterations = 1000; // 12000;
    const double zoomFactor = 0.9;
    const double panFactorBase = 20.0; // 0.1;

    while (!WindowShouldClose()) {
        // Controles de zoom
        if (IsKeyDown(KEY_UP)) scale *= zoomFactor;
        if (IsKeyDown(KEY_DOWN)) scale /= zoomFactor;

        // Controles de movimentação
        double panFactor = panFactorBase * scale;
        if (IsKeyDown(KEY_D)) centerX += panFactor;
        if (IsKeyDown(KEY_A)) centerX -= panFactor;
        if (IsKeyDown(KEY_W)) centerY -= panFactor; // Movimenta para cima com W
        if (IsKeyDown(KEY_S)) centerY += panFactor; // Movimenta para baixo com S

        // Gera o fractal com os novos valores de centro e escala
        burning_ship_kernel<<<block_dim, thread_dim>>>(pixels, width, height, centerX, centerY, scale, max_iterations);
        hipDeviceSynchronize();

        // Cria uma imagem e textura a partir do array de pixels
        Image image = GenImageColor(width, height, BLACK);
        memcpy(image.data, pixels, width * height * sizeof(Color));
        Texture2D texture = LoadTextureFromImage(image);
        UnloadImage(image); // Descarrega a imagem já que agora temos a textura

        BeginDrawing();
        ClearBackground(RAYWHITE);
        DrawTexture(texture, 0, 0, WHITE);
        EndDrawing();

        UnloadTexture(texture); // Descarrega a textura para regenerar na próxima iteração
    }

    hipFree(pixels); // Libera a memória CUDA
    CloseWindow(); // Fecha a janela e o contexto OpenGL

    return 0;
}
